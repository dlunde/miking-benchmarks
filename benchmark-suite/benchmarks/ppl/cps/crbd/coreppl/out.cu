#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include <math.h>
#include "inference/smc/smc.cuh"
#include <stdint.h>
#include <stdio.h>
typedef struct Tree Tree;
typedef struct Rec {double age; Tree (*left); Tree (*right);} Rec;
typedef struct Rec1 {double age;} Rec1;
enum constrs {Node, Leaf};
typedef struct Tree {enum constrs constr; union {Rec (*Node); Rec1 Leaf;};} Tree;
INIT_MODEL_STACK()
struct GLOBAL {double ret; double lambda; double mu;};
struct STACK_init {pplFunc_t ra; double (*retValLoc); Rec (*root);};
struct STACK_walk {pplFunc_t ra; double nodeAge; Rec (*n); double rho; double inf; double lambda; double mu; Tree (*node); double parentAge;};
BBLOCK_DECLARE(start);
BBLOCK_DECLARE(end);
int64_t countLeaves(Tree (*));
BBLOCK_HELPER_DECLARE(STOCH_countLeaves, int64_t, Tree (*));
double getAge(Tree (*));
BBLOCK_HELPER_DECLARE(STOCH_getAge, double, Tree (*));
double externalLog(double);
BBLOCK_HELPER_DECLARE(STOCH_externalLog, double, double);
double log1(double);
BBLOCK_HELPER_DECLARE(STOCH_log, double, double);
double work(double, int64_t);
BBLOCK_HELPER_DECLARE(STOCH_work, double, double, int64_t);
double logFactorial(int64_t);
BBLOCK_HELPER_DECLARE(STOCH_logFactorial, double, int64_t);
BBLOCK_HELPER_DECLARE(survives, char, double, double, double, double);
BBLOCK_HELPER_DECLARE(simHiddenSpeciation, void, double, double, double, double, double, double);
BBLOCK_DECLARE(bblock);
BBLOCK_DECLARE(bblock1);
BBLOCK_DECLARE(bblock2);
BBLOCK_DECLARE(walk);
BBLOCK_DECLARE(bblock3);
BBLOCK_DECLARE(bblock4);
BBLOCK_DECLARE(init);
BBLOCK(start, {
  struct GLOBAL (*global) = (( struct GLOBAL (*) ) (PSTATE.stack));
  ((PSTATE.stackPtr) = (sizeof(struct GLOBAL)));
  struct STACK_init (*callsf) = (( struct STACK_init (*) ) ((PSTATE.stack) + (( uintptr_t ) (PSTATE.stackPtr))));
  ((callsf->ra) = end);
  ((callsf->retValLoc) = (( double (*) ) ((( char (*) ) (&(global->ret))) - (PSTATE.stack))));
  ((PSTATE.stackPtr) = ((PSTATE.stackPtr) + (sizeof(struct STACK_init))));
  BBLOCK_JUMP(init, NULL);
})
BBLOCK(end, {
  (NEXT = NULL);
})
int64_t countLeaves(Tree (*tree)) {
  if (((tree->constr) == Node)) {
    Rec (*r) = (tree->Node);
    Tree (*t);
    Tree (*X) = (r->left);
    (t = X);
    int64_t t1;
    (t1 = countLeaves(t));
    Tree (*t2);
    Tree (*X1) = (r->right);
    (t2 = X1);
    int64_t t3;
    (t3 = countLeaves(t2));
    int64_t t4;
    (t4 = (t1 + t3));
    return t4;
  } else {
    return 1;
  }
}
BBLOCK_HELPER(STOCH_countLeaves, {
  struct GLOBAL (*global) = (( struct GLOBAL (*) ) (PSTATE.stack));
  if (((tree->constr) == Node)) {
    Rec (*r) = (tree->Node);
    Tree (*t);
    Tree (*X) = (r->left);
    (t = X);
    int64_t t1;
    (t1 = BBLOCK_CALL(STOCH_countLeaves, t));
    Tree (*t2);
    Tree (*X1) = (r->right);
    (t2 = X1);
    int64_t t3;
    (t3 = BBLOCK_CALL(STOCH_countLeaves, t2));
    int64_t t4;
    (t4 = (t1 + t3));
    return t4;
  } else {
    return 1;
  }
}, int64_t, Tree (*tree))
double getAge(Tree (*n1)) {
  if (((n1->constr) == Node)) {
    Rec (*r1) = (n1->Node);
    double X2 = (r1->age);
    return X2;
  } else {
    if (((n1->constr) == Leaf)) {
      Rec1 r2 = (n1->Leaf);
      double X3 = (r2.age);
      return X3;
    } else {
      ;
    }
  }
}
BBLOCK_HELPER(STOCH_getAge, {
  struct GLOBAL (*global) = (( struct GLOBAL (*) ) (PSTATE.stack));
  if (((n1->constr) == Node)) {
    Rec (*r1) = (n1->Node);
    double X2 = (r1->age);
    return X2;
  } else {
    if (((n1->constr) == Leaf)) {
      Rec1 r2 = (n1->Leaf);
      double X3 = (r2.age);
      return X3;
    } else {
      ;
    }
  }
}, double, Tree (*n1))
BBLOCK_DATA_MANAGED_SINGLE(t5, Rec1)
BBLOCK_DATA_MANAGED(t6, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t7, Rec1)
BBLOCK_DATA_MANAGED(t8, Tree, 1)
BBLOCK_DATA_MANAGED(t9, Rec, 1)
BBLOCK_DATA_MANAGED(t10, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t11, Rec1)
BBLOCK_DATA_MANAGED(t12, Tree, 1)
BBLOCK_DATA_MANAGED(t13, Rec, 1)
BBLOCK_DATA_MANAGED(t14, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t15, Rec1)
BBLOCK_DATA_MANAGED(t16, Tree, 1)
BBLOCK_DATA_MANAGED(t17, Rec, 1)
BBLOCK_DATA_MANAGED(t18, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t19, Rec1)
BBLOCK_DATA_MANAGED(t20, Tree, 1)
BBLOCK_DATA_MANAGED(t21, Rec, 1)
BBLOCK_DATA_MANAGED(t22, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t23, Rec1)
BBLOCK_DATA_MANAGED(t24, Tree, 1)
BBLOCK_DATA_MANAGED(t25, Rec, 1)
BBLOCK_DATA_MANAGED(t26, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t27, Rec1)
BBLOCK_DATA_MANAGED(t28, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t29, Rec1)
BBLOCK_DATA_MANAGED(t30, Tree, 1)
BBLOCK_DATA_MANAGED(t31, Rec, 1)
BBLOCK_DATA_MANAGED(t32, Tree, 1)
BBLOCK_DATA_MANAGED(t33, Rec, 1)
BBLOCK_DATA_MANAGED(t34, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t35, Rec1)
BBLOCK_DATA_MANAGED(t36, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t37, Rec1)
BBLOCK_DATA_MANAGED(t38, Tree, 1)
BBLOCK_DATA_MANAGED(t39, Rec, 1)
BBLOCK_DATA_MANAGED(t40, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t41, Rec1)
BBLOCK_DATA_MANAGED(t42, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t43, Rec1)
BBLOCK_DATA_MANAGED(t44, Tree, 1)
BBLOCK_DATA_MANAGED(t45, Rec, 1)
BBLOCK_DATA_MANAGED(t46, Tree, 1)
BBLOCK_DATA_MANAGED(t47, Rec, 1)
BBLOCK_DATA_MANAGED(t48, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t49, Rec1)
BBLOCK_DATA_MANAGED(t50, Tree, 1)
BBLOCK_DATA_MANAGED(t51, Rec, 1)
BBLOCK_DATA_MANAGED(t52, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t53, Rec1)
BBLOCK_DATA_MANAGED(t54, Tree, 1)
BBLOCK_DATA_MANAGED(t55, Rec, 1)
BBLOCK_DATA_MANAGED(t56, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t57, Rec1)
BBLOCK_DATA_MANAGED(t58, Tree, 1)
BBLOCK_DATA_MANAGED(t59, Rec, 1)
BBLOCK_DATA_MANAGED(t60, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t61, Rec1)
BBLOCK_DATA_MANAGED(t62, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t63, Rec1)
BBLOCK_DATA_MANAGED(t64, Tree, 1)
BBLOCK_DATA_MANAGED(t65, Rec, 1)
BBLOCK_DATA_MANAGED(t66, Tree, 1)
BBLOCK_DATA_MANAGED(t67, Rec, 1)
BBLOCK_DATA_MANAGED(t68, Tree, 1)
BBLOCK_DATA_MANAGED(t69, Rec, 1)
BBLOCK_DATA_MANAGED(t70, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t71, Rec1)
BBLOCK_DATA_MANAGED(t72, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t73, Rec1)
BBLOCK_DATA_MANAGED(t74, Tree, 1)
BBLOCK_DATA_MANAGED(t75, Rec, 1)
BBLOCK_DATA_MANAGED(t76, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t77, Rec1)
BBLOCK_DATA_MANAGED(t78, Tree, 1)
BBLOCK_DATA_MANAGED(t79, Rec, 1)
BBLOCK_DATA_MANAGED(t80, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t81, Rec1)
BBLOCK_DATA_MANAGED(t82, Tree, 1)
BBLOCK_DATA_MANAGED(t83, Rec, 1)
BBLOCK_DATA_MANAGED(t84, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t85, Rec1)
BBLOCK_DATA_MANAGED(t86, Tree, 1)
BBLOCK_DATA_MANAGED(t87, Rec, 1)
BBLOCK_DATA_MANAGED(t88, Tree, 1)
BBLOCK_DATA_MANAGED(t89, Rec, 1)
BBLOCK_DATA_MANAGED(t90, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t91, Rec1)
BBLOCK_DATA_MANAGED(t92, Tree, 1)
BBLOCK_DATA_MANAGED(t93, Rec, 1)
BBLOCK_DATA_MANAGED(t94, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t95, Rec1)
BBLOCK_DATA_MANAGED(t96, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t97, Rec1)
BBLOCK_DATA_MANAGED(t98, Tree, 1)
BBLOCK_DATA_MANAGED(t99, Rec, 1)
BBLOCK_DATA_MANAGED(t100, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t101, Rec1)
BBLOCK_DATA_MANAGED(t102, Tree, 1)
BBLOCK_DATA_MANAGED(t103, Rec, 1)
BBLOCK_DATA_MANAGED(t104, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t105, Rec1)
BBLOCK_DATA_MANAGED(t106, Tree, 1)
BBLOCK_DATA_MANAGED(t107, Rec, 1)
BBLOCK_DATA_MANAGED(t108, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t109, Rec1)
BBLOCK_DATA_MANAGED(t110, Tree, 1)
BBLOCK_DATA_MANAGED(t111, Rec, 1)
BBLOCK_DATA_MANAGED(t112, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t113, Rec1)
BBLOCK_DATA_MANAGED(t114, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t115, Rec1)
BBLOCK_DATA_MANAGED(t116, Tree, 1)
BBLOCK_DATA_MANAGED(t117, Rec, 1)
BBLOCK_DATA_MANAGED(t118, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t119, Rec1)
BBLOCK_DATA_MANAGED(t120, Tree, 1)
BBLOCK_DATA_MANAGED(t121, Rec, 1)
BBLOCK_DATA_MANAGED(t122, Tree, 1)
BBLOCK_DATA_MANAGED(t123, Rec, 1)
BBLOCK_DATA_MANAGED(t124, Tree, 1)
BBLOCK_DATA_MANAGED(t125, Rec, 1)
BBLOCK_DATA_MANAGED(t126, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t127, Rec1)
BBLOCK_DATA_MANAGED(t128, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t129, Rec1)
BBLOCK_DATA_MANAGED(t130, Tree, 1)
BBLOCK_DATA_MANAGED(t131, Rec, 1)
BBLOCK_DATA_MANAGED(t132, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t133, Rec1)
BBLOCK_DATA_MANAGED(t134, Tree, 1)
BBLOCK_DATA_MANAGED(t135, Rec, 1)
BBLOCK_DATA_MANAGED(t136, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t137, Rec1)
BBLOCK_DATA_MANAGED(t138, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t139, Rec1)
BBLOCK_DATA_MANAGED(t140, Tree, 1)
BBLOCK_DATA_MANAGED(t141, Rec, 1)
BBLOCK_DATA_MANAGED(t142, Tree, 1)
BBLOCK_DATA_MANAGED(t143, Rec, 1)
BBLOCK_DATA_MANAGED(t144, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t145, Rec1)
BBLOCK_DATA_MANAGED(t146, Tree, 1)
BBLOCK_DATA_MANAGED(t147, Rec, 1)
BBLOCK_DATA_MANAGED(t148, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t149, Rec1)
BBLOCK_DATA_MANAGED(t150, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t151, Rec1)
BBLOCK_DATA_MANAGED(t152, Tree, 1)
BBLOCK_DATA_MANAGED(t153, Rec, 1)
BBLOCK_DATA_MANAGED(t154, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t155, Rec1)
BBLOCK_DATA_MANAGED(t156, Tree, 1)
BBLOCK_DATA_MANAGED(t157, Rec, 1)
BBLOCK_DATA_MANAGED(t158, Tree, 1)
BBLOCK_DATA_MANAGED(t159, Rec, 1)
BBLOCK_DATA_MANAGED(t160, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t161, Rec1)
BBLOCK_DATA_MANAGED(t162, Tree, 1)
BBLOCK_DATA_MANAGED(t163, Rec, 1)
BBLOCK_DATA_MANAGED(t164, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t165, Rec1)
BBLOCK_DATA_MANAGED(t166, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t167, Rec1)
BBLOCK_DATA_MANAGED(t168, Tree, 1)
BBLOCK_DATA_MANAGED(t169, Rec, 1)
BBLOCK_DATA_MANAGED(t170, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t171, Rec1)
BBLOCK_DATA_MANAGED(t172, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t173, Rec1)
BBLOCK_DATA_MANAGED(t174, Tree, 1)
BBLOCK_DATA_MANAGED(t175, Rec, 1)
BBLOCK_DATA_MANAGED(t176, Tree, 1)
BBLOCK_DATA_MANAGED(t177, Rec, 1)
BBLOCK_DATA_MANAGED(t178, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t179, Rec1)
BBLOCK_DATA_MANAGED(t180, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t181, Rec1)
BBLOCK_DATA_MANAGED(t182, Tree, 1)
BBLOCK_DATA_MANAGED(t183, Rec, 1)
BBLOCK_DATA_MANAGED(t184, Tree, 1)
BBLOCK_DATA_MANAGED(t185, Rec, 1)
BBLOCK_DATA_MANAGED(t186, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t187, Rec1)
BBLOCK_DATA_MANAGED(t188, Tree, 1)
BBLOCK_DATA_MANAGED(t189, Rec, 1)
BBLOCK_DATA_MANAGED(t190, Tree, 1)
BBLOCK_DATA_MANAGED(t191, Rec, 1)
BBLOCK_DATA_MANAGED(t192, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t193, Rec1)
BBLOCK_DATA_MANAGED(t194, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t195, Rec1)
BBLOCK_DATA_MANAGED(t196, Tree, 1)
BBLOCK_DATA_MANAGED(t197, Rec, 1)
BBLOCK_DATA_MANAGED(t198, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t199, Rec1)
BBLOCK_DATA_MANAGED(t200, Tree, 1)
BBLOCK_DATA_MANAGED(t201, Rec, 1)
BBLOCK_DATA_MANAGED(t202, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t203, Rec1)
BBLOCK_DATA_MANAGED(t204, Tree, 1)
BBLOCK_DATA_MANAGED(t205, Rec, 1)
BBLOCK_DATA_MANAGED(t206, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t207, Rec1)
BBLOCK_DATA_MANAGED(t208, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(t209, Rec1)
BBLOCK_DATA_MANAGED(t210, Tree, 1)
BBLOCK_DATA_MANAGED(t211, Rec, 1)
BBLOCK_DATA_MANAGED(t212, Tree, 1)
BBLOCK_DATA_MANAGED(t213, Rec, 1)
BBLOCK_DATA_MANAGED(t214, Tree, 1)
BBLOCK_DATA_MANAGED(t215, Rec, 1)
BBLOCK_DATA_MANAGED(t216, Tree, 1)
BBLOCK_DATA_MANAGED(t217, Rec, 1)
BBLOCK_DATA_MANAGED(tree1, Tree, 1)
BBLOCK_DATA_MANAGED_SINGLE(rho, double)
double externalLog(double a1) {
  return log(a1);
}
BBLOCK_HELPER(STOCH_externalLog, {
  struct GLOBAL (*global) = (( struct GLOBAL (*) ) (PSTATE.stack));
  return log(a1);
}, double, double a1)
double log1(double x) {
  double t218;
  (t218 = log(x));
  return t218;
}
BBLOCK_HELPER(STOCH_log, {
  struct GLOBAL (*global) = (( struct GLOBAL (*) ) (PSTATE.stack));
  double t218;
  (t218 = log(x));
  return t218;
}, double, double x)
BBLOCK_DATA_MANAGED_SINGLE(inf, double)
double work(double acc, int64_t n2) {
  char t219;
  (t219 = (n2 > 0));
  if ((t219 == 1)) {
    double t220;
    (t220 = (( double ) n2));
    double t221;
    (t221 = log1(t220));
    double t222;
    (t222 = (t221 + acc));
    int64_t t223;
    (t223 = (n2 - 1));
    double t224;
    (t224 = work(t222, t223));
    return t224;
  } else {
    return acc;
  }
}
BBLOCK_HELPER(STOCH_work, {
  struct GLOBAL (*global) = (( struct GLOBAL (*) ) (PSTATE.stack));
  char t219;
  (t219 = (n2 > 0));
  if ((t219 == 1)) {
    double t220;
    (t220 = (( double ) n2));
    double t221;
    (t221 = BBLOCK_CALL(STOCH_log, t220));
    double t222;
    (t222 = (t221 + acc));
    int64_t t223;
    (t223 = (n2 - 1));
    double t224;
    (t224 = BBLOCK_CALL(STOCH_work, t222, t223));
    return t224;
  } else {
    return acc;
  }
}, double, double acc, int64_t n2)
double logFactorial(int64_t n3) {
  double t225;
  (t225 = work(0., n3));
  return t225;
}
BBLOCK_HELPER(STOCH_logFactorial, {
  struct GLOBAL (*global) = (( struct GLOBAL (*) ) (PSTATE.stack));
  double t225;
  (t225 = BBLOCK_CALL(STOCH_work, 0., n3));
  return t225;
}, double, int64_t n3)
BBLOCK_HELPER(survives, {
  struct GLOBAL (*global) = (( struct GLOBAL (*) ) (PSTATE.stack));
  double t226;
  (t226 = ((global->lambda) + (global->mu)));
  double t227;
  (t227 = (SAMPLE(exponential, t226)));
  double t228;
  (t228 = (tBeg - t227));
  char t229;
  (t229 = (t228 < 0.));
  if ((t229 == 1)) {
    return (SAMPLE(bernoulli, rho));
  } else {
    double t230;
    (t230 = ((global->lambda) + (global->mu)));
    double t231;
    (t231 = ((global->lambda) / t230));
    char t232;
    (t232 = (SAMPLE(bernoulli, t231)));
    if ((t232 == 1)) {
      char t233;
      (t233 = BBLOCK_CALL(survives, rho, (global->lambda), (global->mu), t228));
      if ((t233 == 1)) {
        return 1;
      } else {
        char t234;
        (t234 = BBLOCK_CALL(survives, rho, (global->lambda), (global->mu), t228));
        return t234;
      }
    } else {
      return 0;
    }
  }
}, char, double rho, double lambda, double mu, double tBeg)
BBLOCK_HELPER(simHiddenSpeciation, {
  struct GLOBAL (*global) = (( struct GLOBAL (*) ) (PSTATE.stack));
  double t235;
  (t235 = (SAMPLE(exponential, (global->lambda))));
  double t236;
  (t236 = (tBeg1 - t235));
  char t237;
  (t237 = (t236 > nodeAge));
  if ((t237 == 1)) {
    char t238;
    (t238 = BBLOCK_CALL(survives, rho, (global->lambda), (global->mu), t236));
    if ((t238 == 1)) {
      double t239;
      (t239 = (-inf));
      (WEIGHT(t239));
    } else {
      double t240;
      (t240 = BBLOCK_CALL(STOCH_log, 2.));
      (WEIGHT(t240));
      BBLOCK_CALL(simHiddenSpeciation, rho, inf, (global->lambda), (global->mu), nodeAge, t236);
    }
  } else {
    ;
  }
}, void, double rho, double inf, double lambda, double mu, double nodeAge, double tBeg1)
BBLOCK(bblock, {
  struct GLOBAL (*global) = (( struct GLOBAL (*) ) (PSTATE.stack));
  struct STACK_walk (*sf) = (( struct STACK_walk (*) ) ((PSTATE.stack) + (( uintptr_t ) ((PSTATE.stackPtr) - (sizeof(struct STACK_walk))))));
  ((PSTATE.stackPtr) = ((PSTATE.stackPtr) - (sizeof(struct STACK_walk))));
  BBLOCK_JUMP((sf->ra), NULL);
})
BBLOCK(bblock1, {
  struct GLOBAL (*global) = (( struct GLOBAL (*) ) (PSTATE.stack));
  struct STACK_walk (*sf) = (( struct STACK_walk (*) ) ((PSTATE.stack) + (( uintptr_t ) ((PSTATE.stackPtr) - (sizeof(struct STACK_walk))))));
  Tree (*t241);
  Tree (*X4) = ((sf->n)->right);
  (t241 = X4);
  struct STACK_walk (*callsf) = (( struct STACK_walk (*) ) ((PSTATE.stack) + (( uintptr_t ) (PSTATE.stackPtr))));
  ((callsf->ra) = bblock);
  ((callsf->rho) = (sf->rho));
  ((callsf->inf) = (sf->inf));
  ((callsf->lambda) = (sf->lambda));
  ((callsf->mu) = (sf->mu));
  ((callsf->node) = t241);
  ((callsf->parentAge) = (sf->nodeAge));
  ((PSTATE.stackPtr) = ((PSTATE.stackPtr) + (sizeof(struct STACK_walk))));
  BBLOCK_JUMP(walk, NULL);
})
BBLOCK(bblock2, {
  struct GLOBAL (*global) = (( struct GLOBAL (*) ) (PSTATE.stack));
  struct STACK_walk (*sf) = (( struct STACK_walk (*) ) ((PSTATE.stack) + (( uintptr_t ) ((PSTATE.stackPtr) - (sizeof(struct STACK_walk))))));
  Tree (*t242);
  Tree (*X5) = ((sf->n)->left);
  (t242 = X5);
  struct STACK_walk (*callsf) = (( struct STACK_walk (*) ) ((PSTATE.stack) + (( uintptr_t ) (PSTATE.stackPtr))));
  ((callsf->ra) = bblock1);
  ((callsf->rho) = (sf->rho));
  ((callsf->inf) = (sf->inf));
  ((callsf->lambda) = (sf->lambda));
  ((callsf->mu) = (sf->mu));
  ((callsf->node) = t242);
  ((callsf->parentAge) = (sf->nodeAge));
  ((PSTATE.stackPtr) = ((PSTATE.stackPtr) + (sizeof(struct STACK_walk))));
  BBLOCK_JUMP(walk, NULL);
})
BBLOCK(walk, {
  struct GLOBAL (*global) = (( struct GLOBAL (*) ) (PSTATE.stack));
  struct STACK_walk (*sf) = (( struct STACK_walk (*) ) ((PSTATE.stack) + (( uintptr_t ) ((PSTATE.stackPtr) - (sizeof(struct STACK_walk))))));
  ((sf->nodeAge) = BBLOCK_CALL(STOCH_getAge, (sf->node)));
  BBLOCK_CALL(simHiddenSpeciation, (sf->rho), (sf->inf), (sf->lambda), (sf->mu), (sf->nodeAge), (sf->parentAge));
  double t243;
  (t243 = ((sf->parentAge) - (sf->nodeAge)));
  double t244;
  (t244 = ((sf->mu) * t243));
  (OBSERVE(poisson, t244, 0));
  if ((((sf->node)->constr) == Node)) {
    ((sf->n) = ((sf->node)->Node));
    (OBSERVE(exponential, (sf->lambda), 0.));
    (NEXT = bblock2);
  } else {
    if ((((sf->node)->constr) == Leaf)) {
      (OBSERVE(bernoulli, (sf->rho), 1));
      ((PSTATE.stackPtr) = ((PSTATE.stackPtr) - (sizeof(struct STACK_walk))));
      (NEXT = (sf->ra));
    } else {
      ;
      ((PSTATE.stackPtr) = ((PSTATE.stackPtr) - (sizeof(struct STACK_walk))));
      BBLOCK_JUMP((sf->ra), NULL);
    }
  }
})
BBLOCK_DATA_MANAGED_SINGLE(numLeaves, int64_t)
BBLOCK_DATA_MANAGED_SINGLE(t245, double)
BBLOCK_DATA_MANAGED_SINGLE(t246, double)
BBLOCK_DATA_MANAGED_SINGLE(t247, double)
BBLOCK_DATA_MANAGED_SINGLE(t248, double)
BBLOCK_DATA_MANAGED_SINGLE(t249, double)
BBLOCK_DATA_MANAGED_SINGLE(t250, double)
BBLOCK(bblock3, {
  struct GLOBAL (*global) = (( struct GLOBAL (*) ) (PSTATE.stack));
  struct STACK_init (*sf) = (( struct STACK_init (*) ) ((PSTATE.stack) + (( uintptr_t ) ((PSTATE.stackPtr) - (sizeof(struct STACK_init))))));
  ((*(( double (*) ) ((PSTATE.stack) + (( uintptr_t ) (sf->retValLoc))))) = (global->lambda));
  ((PSTATE.stackPtr) = ((PSTATE.stackPtr) - (sizeof(struct STACK_init))));
  BBLOCK_JUMP((sf->ra), NULL);
})
BBLOCK(bblock4, {
  struct GLOBAL (*global) = (( struct GLOBAL (*) ) (PSTATE.stack));
  struct STACK_init (*sf) = (( struct STACK_init (*) ) ((PSTATE.stack) + (( uintptr_t ) ((PSTATE.stackPtr) - (sizeof(struct STACK_init))))));
  Tree (*t251);
  Tree (*X6) = ((sf->root)->right);
  (t251 = X6);
  double t252;
  double X7 = ((sf->root)->age);
  (t252 = X7);
  struct STACK_walk (*callsf) = (( struct STACK_walk (*) ) ((PSTATE.stack) + (( uintptr_t ) (PSTATE.stackPtr))));
  ((callsf->ra) = bblock3);
  ((callsf->rho) = rho);
  ((callsf->inf) = inf);
  ((callsf->lambda) = (global->lambda));
  ((callsf->mu) = (global->mu));
  ((callsf->node) = t251);
  ((callsf->parentAge) = t252);
  ((PSTATE.stackPtr) = ((PSTATE.stackPtr) + (sizeof(struct STACK_walk))));
  BBLOCK_JUMP(walk, NULL);
})
BBLOCK(init, {
  struct GLOBAL (*global) = (( struct GLOBAL (*) ) (PSTATE.stack));
  struct STACK_init (*sf) = (( struct STACK_init (*) ) ((PSTATE.stack) + (( uintptr_t ) ((PSTATE.stackPtr) - (sizeof(struct STACK_init))))));
  ((global->lambda) = (SAMPLE(gamma, 1., 1.)));
  ((global->mu) = (SAMPLE(gamma, 1., 0.5)));
  (WEIGHT(t250));
  if (((tree1->constr) == Node)) {
    ((sf->root) = (tree1->Node));
    Tree (*t253);
    Tree (*X8) = ((sf->root)->left);
    (t253 = X8);
    double t254;
    double X9 = ((sf->root)->age);
    (t254 = X9);
    struct STACK_walk (*callsf) = (( struct STACK_walk (*) ) ((PSTATE.stack) + (( uintptr_t ) (PSTATE.stackPtr))));
    ((callsf->ra) = bblock4);
    ((callsf->rho) = rho);
    ((callsf->inf) = inf);
    ((callsf->lambda) = (global->lambda));
    ((callsf->mu) = (global->mu));
    ((callsf->node) = t253);
    ((callsf->parentAge) = t254);
    ((PSTATE.stackPtr) = ((PSTATE.stackPtr) + (sizeof(struct STACK_walk))));
    BBLOCK_JUMP(walk, NULL);
  } else {
    ;
    ((PSTATE.stackPtr) = ((PSTATE.stackPtr) - (sizeof(struct STACK_init))));
    BBLOCK_JUMP((sf->ra), NULL);
  }
})
CALLBACK(callback, {
  int i = 0;
  while ((i < N)) {
    struct GLOBAL (*global) = (( struct GLOBAL (*) ) ((PSTATES[i]).stack));
    printf("%f %f\n", (global->ret), (WEIGHTS[i]));
    (i = (i + 1));
  }
})
MAIN({
  ((t5.age) = 0.);
  ((t6->constr) = Leaf);
  ((t6->Leaf) = t5);
  ((t7.age) = 0.);
  ((t8->constr) = Leaf);
  ((t8->Leaf) = t7);
  ((t9->age) = 1.900561313);
  ((t9->left) = t8);
  ((t9->right) = t6);
  ((t10->constr) = Node);
  ((t10->Node) = t9);
  ((t11.age) = 0.);
  ((t12->constr) = Leaf);
  ((t12->Leaf) = t11);
  ((t13->age) = 3.100150132);
  ((t13->left) = t12);
  ((t13->right) = t10);
  ((t14->constr) = Node);
  ((t14->Node) = t13);
  ((t15.age) = 0.);
  ((t16->constr) = Leaf);
  ((t16->Leaf) = t15);
  ((t17->age) = 6.043650727);
  ((t17->left) = t16);
  ((t17->right) = t14);
  ((t18->constr) = Node);
  ((t18->Node) = t17);
  ((t19.age) = 0.);
  ((t20->constr) = Leaf);
  ((t20->Leaf) = t19);
  ((t21->age) = 12.38252513);
  ((t21->left) = t20);
  ((t21->right) = t18);
  ((t22->constr) = Node);
  ((t22->Node) = t21);
  ((t23.age) = 0.);
  ((t24->constr) = Leaf);
  ((t24->Leaf) = t23);
  ((t25->age) = 12.61785812);
  ((t25->left) = t24);
  ((t25->right) = t22);
  ((t26->constr) = Node);
  ((t26->Node) = t25);
  ((t27.age) = 0.);
  ((t28->constr) = Leaf);
  ((t28->Leaf) = t27);
  ((t29.age) = 0.);
  ((t30->constr) = Leaf);
  ((t30->Leaf) = t29);
  ((t31->age) = 11.15685875);
  ((t31->left) = t30);
  ((t31->right) = t28);
  ((t32->constr) = Node);
  ((t32->Node) = t31);
  ((t33->age) = 15.396725736);
  ((t33->left) = t32);
  ((t33->right) = t26);
  ((t34->constr) = Node);
  ((t34->Node) = t33);
  ((t35.age) = 0.);
  ((t36->constr) = Leaf);
  ((t36->Leaf) = t35);
  ((t37.age) = 0.);
  ((t38->constr) = Leaf);
  ((t38->Leaf) = t37);
  ((t39->age) = 1.04896206);
  ((t39->left) = t38);
  ((t39->right) = t36);
  ((t40->constr) = Node);
  ((t40->Node) = t39);
  ((t41.age) = 0.);
  ((t42->constr) = Leaf);
  ((t42->Leaf) = t41);
  ((t43.age) = 0.);
  ((t44->constr) = Leaf);
  ((t44->Leaf) = t43);
  ((t45->age) = 0.9841688636);
  ((t45->left) = t44);
  ((t45->right) = t42);
  ((t46->constr) = Node);
  ((t46->Node) = t45);
  ((t47->age) = 1.7140599232);
  ((t47->left) = t46);
  ((t47->right) = t40);
  ((t48->constr) = Node);
  ((t48->Node) = t47);
  ((t49.age) = 0.);
  ((t50->constr) = Leaf);
  ((t50->Leaf) = t49);
  ((t51->age) = 3.786162534);
  ((t51->left) = t50);
  ((t51->right) = t48);
  ((t52->constr) = Node);
  ((t52->Node) = t51);
  ((t53.age) = 0.);
  ((t54->constr) = Leaf);
  ((t54->Leaf) = t53);
  ((t55->age) = 8.788450495);
  ((t55->left) = t54);
  ((t55->right) = t52);
  ((t56->constr) = Node);
  ((t56->Node) = t55);
  ((t57.age) = 0.);
  ((t58->constr) = Leaf);
  ((t58->Leaf) = t57);
  ((t59->age) = 11.05846217);
  ((t59->left) = t58);
  ((t59->right) = t56);
  ((t60->constr) = Node);
  ((t60->Node) = t59);
  ((t61.age) = 0.);
  ((t62->constr) = Leaf);
  ((t62->Leaf) = t61);
  ((t63.age) = 0.);
  ((t64->constr) = Leaf);
  ((t64->Leaf) = t63);
  ((t65->age) = 8.614086751);
  ((t65->left) = t64);
  ((t65->right) = t62);
  ((t66->constr) = Node);
  ((t66->Node) = t65);
  ((t67->age) = 15.008504768);
  ((t67->left) = t66);
  ((t67->right) = t60);
  ((t68->constr) = Node);
  ((t68->Node) = t67);
  ((t69->age) = 16.828404506);
  ((t69->left) = t68);
  ((t69->right) = t34);
  ((t70->constr) = Node);
  ((t70->Node) = t69);
  ((t71.age) = 0.);
  ((t72->constr) = Leaf);
  ((t72->Leaf) = t71);
  ((t73.age) = 0.);
  ((t74->constr) = Leaf);
  ((t74->Leaf) = t73);
  ((t75->age) = 4.220057646);
  ((t75->left) = t74);
  ((t75->right) = t72);
  ((t76->constr) = Node);
  ((t76->Node) = t75);
  ((t77.age) = 0.);
  ((t78->constr) = Leaf);
  ((t78->Leaf) = t77);
  ((t79->age) = 8.451051062);
  ((t79->left) = t78);
  ((t79->right) = t76);
  ((t80->constr) = Node);
  ((t80->Node) = t79);
  ((t81.age) = 0.);
  ((t82->constr) = Leaf);
  ((t82->Leaf) = t81);
  ((t83->age) = 11.54072627);
  ((t83->left) = t82);
  ((t83->right) = t80);
  ((t84->constr) = Node);
  ((t84->Node) = t83);
  ((t85.age) = 0.);
  ((t86->constr) = Leaf);
  ((t86->Leaf) = t85);
  ((t87->age) = 15.28839572);
  ((t87->left) = t86);
  ((t87->right) = t84);
  ((t88->constr) = Node);
  ((t88->Node) = t87);
  ((t89->age) = 20.368109703);
  ((t89->left) = t88);
  ((t89->right) = t70);
  ((t90->constr) = Node);
  ((t90->Node) = t89);
  ((t91.age) = 0.);
  ((t92->constr) = Leaf);
  ((t92->Leaf) = t91);
  ((t93->age) = 23.74299959);
  ((t93->left) = t92);
  ((t93->right) = t90);
  ((t94->constr) = Node);
  ((t94->Node) = t93);
  ((t95.age) = 0.);
  ((t96->constr) = Leaf);
  ((t96->Leaf) = t95);
  ((t97.age) = 0.);
  ((t98->constr) = Leaf);
  ((t98->Leaf) = t97);
  ((t99->age) = 6.306427821);
  ((t99->left) = t98);
  ((t99->right) = t96);
  ((t100->constr) = Node);
  ((t100->Node) = t99);
  ((t101.age) = 0.);
  ((t102->constr) = Leaf);
  ((t102->Leaf) = t101);
  ((t103->age) = 9.40050129);
  ((t103->left) = t102);
  ((t103->right) = t100);
  ((t104->constr) = Node);
  ((t104->Node) = t103);
  ((t105.age) = 0.);
  ((t106->constr) = Leaf);
  ((t106->Leaf) = t105);
  ((t107->age) = 13.85876825);
  ((t107->left) = t106);
  ((t107->right) = t104);
  ((t108->constr) = Node);
  ((t108->Node) = t107);
  ((t109.age) = 0.);
  ((t110->constr) = Leaf);
  ((t110->Leaf) = t109);
  ((t111->age) = 20.68766993);
  ((t111->left) = t110);
  ((t111->right) = t108);
  ((t112->constr) = Node);
  ((t112->Node) = t111);
  ((t113.age) = 0.);
  ((t114->constr) = Leaf);
  ((t114->Leaf) = t113);
  ((t115.age) = 0.);
  ((t116->constr) = Leaf);
  ((t116->Leaf) = t115);
  ((t117->age) = 4.534421013);
  ((t117->left) = t116);
  ((t117->right) = t114);
  ((t118->constr) = Node);
  ((t118->Node) = t117);
  ((t119.age) = 0.);
  ((t120->constr) = Leaf);
  ((t120->Leaf) = t119);
  ((t121->age) = 12.46869821);
  ((t121->left) = t120);
  ((t121->right) = t118);
  ((t122->constr) = Node);
  ((t122->Node) = t121);
  ((t123->age) = 22.82622451);
  ((t123->left) = t122);
  ((t123->right) = t112);
  ((t124->constr) = Node);
  ((t124->Node) = t123);
  ((t125->age) = 32.145876657);
  ((t125->left) = t124);
  ((t125->right) = t94);
  ((t126->constr) = Node);
  ((t126->Node) = t125);
  ((t127.age) = 0.);
  ((t128->constr) = Leaf);
  ((t128->Leaf) = t127);
  ((t129.age) = 0.);
  ((t130->constr) = Leaf);
  ((t130->Leaf) = t129);
  ((t131->age) = 1.962579854);
  ((t131->left) = t130);
  ((t131->right) = t128);
  ((t132->constr) = Node);
  ((t132->Node) = t131);
  ((t133.age) = 0.);
  ((t134->constr) = Leaf);
  ((t134->Leaf) = t133);
  ((t135->age) = 3.732932004);
  ((t135->left) = t134);
  ((t135->right) = t132);
  ((t136->constr) = Node);
  ((t136->Node) = t135);
  ((t137.age) = 0.);
  ((t138->constr) = Leaf);
  ((t138->Leaf) = t137);
  ((t139.age) = 0.);
  ((t140->constr) = Leaf);
  ((t140->Leaf) = t139);
  ((t141->age) = 0.6302632958);
  ((t141->left) = t140);
  ((t141->right) = t138);
  ((t142->constr) = Node);
  ((t142->Node) = t141);
  ((t143->age) = 5.5933070698);
  ((t143->left) = t142);
  ((t143->right) = t136);
  ((t144->constr) = Node);
  ((t144->Node) = t143);
  ((t145.age) = 0.);
  ((t146->constr) = Leaf);
  ((t146->Leaf) = t145);
  ((t147->age) = 6.096453021);
  ((t147->left) = t146);
  ((t147->right) = t144);
  ((t148->constr) = Node);
  ((t148->Node) = t147);
  ((t149.age) = 0.);
  ((t150->constr) = Leaf);
  ((t150->Leaf) = t149);
  ((t151.age) = 0.);
  ((t152->constr) = Leaf);
  ((t152->Leaf) = t151);
  ((t153->age) = 1.519406055);
  ((t153->left) = t152);
  ((t153->right) = t150);
  ((t154->constr) = Node);
  ((t154->Node) = t153);
  ((t155.age) = 0.);
  ((t156->constr) = Leaf);
  ((t156->Leaf) = t155);
  ((t157->age) = 4.987038163);
  ((t157->left) = t156);
  ((t157->right) = t154);
  ((t158->constr) = Node);
  ((t158->Node) = t157);
  ((t159->age) = 8.265483252);
  ((t159->left) = t158);
  ((t159->right) = t148);
  ((t160->constr) = Node);
  ((t160->Node) = t159);
  ((t161.age) = 0.);
  ((t162->constr) = Leaf);
  ((t162->Leaf) = t161);
  ((t163->age) = 10.86835485);
  ((t163->left) = t162);
  ((t163->right) = t160);
  ((t164->constr) = Node);
  ((t164->Node) = t163);
  ((t165.age) = 0.);
  ((t166->constr) = Leaf);
  ((t166->Leaf) = t165);
  ((t167.age) = 0.);
  ((t168->constr) = Leaf);
  ((t168->Leaf) = t167);
  ((t169->age) = 5.054547857);
  ((t169->left) = t168);
  ((t169->right) = t166);
  ((t170->constr) = Node);
  ((t170->Node) = t169);
  ((t171.age) = 0.);
  ((t172->constr) = Leaf);
  ((t172->Leaf) = t171);
  ((t173.age) = 0.);
  ((t174->constr) = Leaf);
  ((t174->Leaf) = t173);
  ((t175->age) = 3.151799953);
  ((t175->left) = t174);
  ((t175->right) = t172);
  ((t176->constr) = Node);
  ((t176->Node) = t175);
  ((t177->age) = 6.284896357);
  ((t177->left) = t176);
  ((t177->right) = t170);
  ((t178->constr) = Node);
  ((t178->Node) = t177);
  ((t179.age) = 0.);
  ((t180->constr) = Leaf);
  ((t180->Leaf) = t179);
  ((t181.age) = 0.);
  ((t182->constr) = Leaf);
  ((t182->Leaf) = t181);
  ((t183->age) = 3.934203877);
  ((t183->left) = t182);
  ((t183->right) = t180);
  ((t184->constr) = Node);
  ((t184->Node) = t183);
  ((t185->age) = 7.815689971);
  ((t185->left) = t184);
  ((t185->right) = t178);
  ((t186->constr) = Node);
  ((t186->Node) = t185);
  ((t187.age) = 0.);
  ((t188->constr) = Leaf);
  ((t188->Leaf) = t187);
  ((t189->age) = 10.32243059);
  ((t189->left) = t188);
  ((t189->right) = t186);
  ((t190->constr) = Node);
  ((t190->Node) = t189);
  ((t191->age) = 12.551924091);
  ((t191->left) = t190);
  ((t191->right) = t164);
  ((t192->constr) = Node);
  ((t192->Node) = t191);
  ((t193.age) = 0.);
  ((t194->constr) = Leaf);
  ((t194->Leaf) = t193);
  ((t195.age) = 0.);
  ((t196->constr) = Leaf);
  ((t196->Leaf) = t195);
  ((t197->age) = 4.788021775);
  ((t197->left) = t196);
  ((t197->right) = t194);
  ((t198->constr) = Node);
  ((t198->Node) = t197);
  ((t199.age) = 0.);
  ((t200->constr) = Leaf);
  ((t200->Leaf) = t199);
  ((t201->age) = 7.595901077);
  ((t201->left) = t200);
  ((t201->right) = t198);
  ((t202->constr) = Node);
  ((t202->Node) = t201);
  ((t203.age) = 0.);
  ((t204->constr) = Leaf);
  ((t204->Leaf) = t203);
  ((t205->age) = 9.436625313);
  ((t205->left) = t204);
  ((t205->right) = t202);
  ((t206->constr) = Node);
  ((t206->Node) = t205);
  ((t207.age) = 0.);
  ((t208->constr) = Leaf);
  ((t208->Leaf) = t207);
  ((t209.age) = 0.);
  ((t210->constr) = Leaf);
  ((t210->Leaf) = t209);
  ((t211->age) = 5.635787971);
  ((t211->left) = t210);
  ((t211->right) = t208);
  ((t212->constr) = Node);
  ((t212->Node) = t211);
  ((t213->age) = 12.344087935);
  ((t213->left) = t212);
  ((t213->right) = t206);
  ((t214->constr) = Node);
  ((t214->Node) = t213);
  ((t215->age) = 13.472886809);
  ((t215->left) = t214);
  ((t215->right) = t192);
  ((t216->constr) = Node);
  ((t216->Node) = t215);
  ((t217->age) = 34.940139089);
  ((t217->left) = t216);
  ((t217->right) = t126);
  ((tree1->constr) = Node);
  ((tree1->Node) = t217);
  (rho = 0.568421052632);
  (inf = (1. / 0.));
  (numLeaves = countLeaves(tree1));
  (t245 = (( double ) numLeaves));
  (t246 = (t245 - 1.));
  (t247 = log1(2.));
  (t248 = (t246 * t247));
  (t249 = logFactorial(numLeaves));
  (t250 = (t248 - t249));
  FIRST_BBLOCK(start);
  SMC(callback);
})